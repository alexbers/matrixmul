#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define SIZE 2048
#define BLOCKSIZE 32

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            printf("Failed to run stmt %s", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

// Compute P = N * M
__global__ void matrixMultiply(double * N, double * M, double * P, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < size) && (col < size)) {
        double sum = 0;
        for(int k = 0; k < size; k++) {
            sum += N[row * size + k] * M[k * size + col];
        }
        P[row * size + col] = sum;
    }
}


int main() {
    struct timeval start, end;

    double *h_N = (double *) malloc(SIZE * SIZE * sizeof(double));
    double *h_M = (double *) malloc(SIZE * SIZE * sizeof(double));
    double *h_P = (double *) malloc(SIZE * SIZE * sizeof(double));

    long i;
    for(i = 0; i < SIZE * SIZE; i++) {
        h_N[i] = 2.0;
        h_M[i] = 2.0;
        h_P[i] = 0.0;
    }

    double *d_N;
    double *d_M;
    double *d_P;

    wbCheck(hipMalloc((void **) &d_N, SIZE * SIZE * sizeof(double)));
    wbCheck(hipMalloc((void **) &d_M, SIZE * SIZE * sizeof(double)));
    wbCheck(hipMalloc((void **) &d_P, SIZE * SIZE * sizeof(double)));

    wbCheck(hipMemcpy(d_N, h_N, SIZE * SIZE * sizeof(double), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(d_M, h_M, SIZE * SIZE * sizeof(double), hipMemcpyHostToDevice));

    dim3 dimGrid(SIZE / BLOCKSIZE, SIZE / BLOCKSIZE,1);
    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);

    gettimeofday(&start, NULL);

    matrixMultiply<<<dimGrid,dimBlock>>>(d_N, d_M, d_P, SIZE);
    hipDeviceSynchronize();

    gettimeofday(&end, NULL);

    wbCheck(hipMemcpy(h_P, d_P, SIZE * SIZE * sizeof(double), hipMemcpyDeviceToHost));

    // time calculation
    if(end.tv_sec < start.tv_sec) {
        printf("You are very unlucky, please, run me again\n");
        return 1;
    }

    double usec_diff = (end.tv_sec - start.tv_sec) +
                       (double)(end.tv_usec - start.tv_usec) / 1000 / 1000;
    double time_spent = (double)(usec_diff);

    printf("Multiplication finished, wallclock: %f sec\n", time_spent);

    return 0;
}
